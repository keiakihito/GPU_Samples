
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {
    printf("Hello World from GPU!\n");
}

int main(void) {
    printf("Hello World from CPU!\n");

    helloFromGPU<<<1, 10>>>();

    // Check for any errors launching the kernel
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "helloFromGPU launch failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching helloFromGPU!\n", cudaStatus);
        return 1;
    }

    return 0;
}
