
#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define CHECK(call){ \
const hipError_t cuda_ret = call; \
if(cuda_ret != hipSuccess){ \
printf("Error: %s:%d,  ", __FILE__, __LINE__ );\
printf("code: %d, reason: %s \n", cuda_ret, hipGetErrorString(cuda_ret));\
exit(-1); \
}\
}

//~~~Helper fundtions
//Input:
//int numOfRow, number of row a matrix
//int numOfClm, number of column a matrix
//const float *ptr_h, dynamic array inital index address
//Process: the function fills up random float number for matrix A and matrix B
//Output: void
void fillUpArray(int numOfRow, int numOfClm, float *ptr_h)
{
    for (int wkr = 0; wkr < numOfRow * numOfClm; wkr++) {
        ptr_h[wkr] = (float)wkr + 1.0;
    }
} // end of fillUpArray

//Input:
//int numOfRow, number of row a matrix
//int numOfClm, number of column a matrix
//const float *ptr_h, dynamic array inital index address
//Process: the function prints the value of array
//Output: void
void printArray(int numOfRow, int numOfClm, const float *ptr_h)
{
    for (int rWkr = 0; rWkr < numOfRow; rWkr++) {
        for (int cWkr = 0; cWkr < numOfClm; cWkr++) {
            printf("%f ", ptr_h[rWkr * numOfClm + cWkr]);
        } // end of inner loop
        printf("\n");
    }// end of outer loop
} // end of printArray

//Input:
//float* CPU_Answer, the initial address of computation result of host function
//float* GPU_Answer, the initial address of computation result of GPU matrix
//unsigned int nRows, number of rows of each matrix
//unsigned int nCols, number of colmuns of each matrix
bool verify(float* CPU_Answer, float* GPU_Answer, unsigned int nRows, unsigned int nCols)
{
    const float epsilon = 10e-6;
    float diff = 0.0f;
    for (int rWkr = 0; rWkr < nRows; rWkr++) {
        for (int cWkr = 0; cWkr < nCols; cWkr++) {
            diff = fabs(CPU_Answer[rWkr*nCols + cWkr] - GPU_Answer[rWkr*nCols + cWkr]);
            if (diff > epsilon) {return false; }
        } // end of inner loop
    }// end of outer loop
    return true;
} // end of verify


//~~~~CUP function~~~~~
//Input:
//int m, number of row matrixA
//int k, number of column matrixA, and number of row matrixB
//int n, number of column matrixB
//Process matrix multiplicatoin C = AB
//Output void.
void basicSgemm_h(int m, int k, int n, const float *A_h, const float *B_h, float* C_h)
{
    float sum = 0.0;
    for (int rWkr = 0; rWkr < m; rWkr++) {
        for(int cWkr = 0; cWkr < n; cWkr++) {
            for(int kWkr = 0; kWkr < k; kWkr++) {
                // printf("\nA_h[%d]", rWkr * k + kWkr);
                // printf("\nB_h[%d]", kWkr * n +cWkr);
                sum += A_h[rWkr * k + kWkr] * B_h[kWkr * n +cWkr];
            } // end of inner loop 2
            // printf("\nrWkr: %d, cWkr: %d", rWkr, cWkr);
            // printf("\nSum: %f", sum);
            // printf("\nC_h[%d]", rWkr*k + cWkr);
            C_h[rWkr*n+ cWkr] = sum;
            sum = 0.0;
        } // end of inner loop 1

    }// end of outer loop
}// end of basicSgemm_h



//~~~~~~CUDA kernel~~~~~~~~~~
//1. 1thread 1 element
//Input:
//int m, number of row matrixA
//int k, number of column matrixA, and number of row matrixB
//int n, number of column matrixB
//Process A CUDA kernel where each thread computes one output matrix element
//Output void.
__global__ void  matrixMulKernel_1thread1element(int m, int k, int n, const float* A_d, const float *B_d, float* C_d)
{
    //Calculate global thread index
    unsigned int rowGlbIdx = blockIdx.y*blockDim.y+threadIdx.y;
    unsigned int clmGlbIdx = blockIdx.x*blockDim.x+threadIdx.x;
     float sum = 0.0f;

    //Boundry condition
    if(rowGlbIdx < m && clmGlbIdx < n) {
        for(unsigned int wkr = 0; wkr < k; wkr++) {
            sum += A_d[rowGlbIdx*k + wkr] * B_d[wkr*k+clmGlbIdx];
        }
        C_d[rowGlbIdx*k + clmGlbIdx] = sum;
    } // end of if

}// end of matrixMulKernel_1thread1element


//2. 1thread 1 row
//Input:
//int m, number of row matrixA
//int k, number of column matrixA, and number of row matrixB
//int n, number of column matrixB
//Process A CUDA kernel where each thread computes one output matrix row.
//Output void
__global__ void matrixMulKernel_1thread1row(int m, int k, int n, const float* A_d, const float *B_d, float* C_d)
{
    //Calculate global thread index
    unsigned int rowGlbIdx = blockIdx.y*blockDim.y+threadIdx.y;
    float sum = 0.0f;

    //Boundry condition
    if(rowGlbIdx < m) {
        for (unsigned int outWkr = 0; outWkr <n; outWkr++) {
            for(unsigned int inWkr = 0; inWkr < k; inWkr++) {
                sum += A_d[rowGlbIdx*k + inWkr] * B_d[inWkr*k + outWkr];
            } // end of inner loop
            C_d[rowGlbIdx*k + outWkr] = sum;
            sum = 0.0f;
        } // end of outer loop

    } // end of if

} // end of matrixMulKernel_1thread1row

//3. 1thread 1 column
//Input:
//int m, number of row matrixA
//int k, number of column matrixA, and number of row matrixB
//int n, number of column matrixB
//Process A CUDA kernel where each thread computes one output matrix row.
//Output void
__global__ void matrixMulKernel_1thread1column(int m, int k, int n, const float* A_d, const float *B_d, float* C_d)
{

    //Calculate global thread index
    unsigned int clmGlbIdx = blockIdx.x*blockDim.x+threadIdx.x;
    float sum = 0.0f;

    //Boundry condition
    if(clmGlbIdx < n) {
        for (unsigned int outWkr = 0; outWkr <m; outWkr++) {
            for(unsigned int inWkr = 0; inWkr < k; inWkr++) {
                sum += A_d[outWkr*k+inWkr] * B_d[inWkr*k + clmGlbIdx];
            } // end of inner loop
            C_d[outWkr*k + clmGlbIdx] = sum;
            sum = 0.0f;
        } // end of outer loop

    } // end of if

} // end of matrixMulKernel_1thread1column


int main(int argc, char** argv)
{
    int m = 3, k = 3, n = 3;

    float* ptrMtxA_h = (float*)malloc((m * k) * sizeof(float));
    printf("\n Matrix A: \n");
    fillUpArray(m, k, ptrMtxA_h);
    printArray(m, k, ptrMtxA_h);

    printf("\n Matrix B: \n");
    float* ptrMtxB_h = (float*)malloc((k * n) * sizeof(float));
    fillUpArray(k, n, ptrMtxB_h);
    printArray(k, n, ptrMtxB_h);

    float* ptrMtxC_h = (float*)malloc((m * n) * sizeof(float));
    float* ptrMtxD_h = (float*)malloc((m * n) * sizeof(float));

    //(0) Calculate Matrix multiplication with CPU functino
    basicSgemm_h(m,k,n, ptrMtxA_h, ptrMtxB_h, ptrMtxC_h);

    //(1) Allocate device memory for arrays x_d, y_d, and z_d.
    float* ptrMtxA_d = NULL;
    float* ptrMtxB_d = NULL;
    float* ptrMtxD_d = NULL;
    CHECK(hipMalloc((void**)&ptrMtxA_d, sizeof(float)*(m * k)));
    CHECK(hipMalloc((void**)&ptrMtxB_d, sizeof(float)*(k * n)));
    CHECK(hipMalloc((void**)&ptrMtxD_d, sizeof(float)*(m * n)));

    //(2) Copy arrays x_h and y_h to device memoery x_d and y_d, respectively.
    CHECK(hipMemcpy(ptrMtxA_d, ptrMtxA_h, sizeof(float)*(m * k), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(ptrMtxB_d, ptrMtxB_h, sizeof(float)*(k * n), hipMemcpyHostToDevice));

    //(3) Call kernel to launch a grid of threads to perform the computation on GPU.
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(ceil((float)n/ blockDim.x), ceil((float)m/blockDim.y),1);

    //2.1
    // matrixMulKernel_1thread1element<<<gridDim, blockDim>>>(m, k, n, ptrMtxA_d, ptrMtxB_d, ptrMtxD_d);

    // 2.2
    // matrixMulKernel_1thread1row<<<gridDim, blockDim>>>(m, k, n, ptrMtxA_d, ptrMtxB_d, ptrMtxD_d);

    //2.3
    matrixMulKernel_1thread1column<<<gridDim, blockDim>>>(m, k, n, ptrMtxA_d, ptrMtxB_d, ptrMtxD_d);


    //(4) Copy the result data from the device memory of array z_d to the host memory of array z_h.
    CHECK(hipMemcpy(ptrMtxD_h, ptrMtxD_d, sizeof(float)*(m*n), hipMemcpyDeviceToHost));
    printf("\n Matrix C: \n");
    printArray(m,n, ptrMtxC_h);
    printf("\n Matrix D: \n");
    printArray(m,n, ptrMtxD_h);

    printf("\nIs Matrix C == Matirx D? : ");
    printf("%d\n",verify(ptrMtxC_h, ptrMtxD_h, m,n));

    //(5) Free device memory of arrays x_d, y_d, and z_d
    CHECK(hipFree(ptrMtxA_d));
    CHECK(hipFree(ptrMtxB_d));
    CHECK(hipFree(ptrMtxD_d));

    // Free host memory of arrays x_h, y_h, and z_h
    free(ptrMtxA_h);
    ptrMtxA_h = NULL;
    free(ptrMtxB_h);
    ptrMtxB_h = NULL;
    free(ptrMtxC_h);
    ptrMtxC_h = NULL;
    free(ptrMtxD_h);
    ptrMtxD_h = NULL;

    return 0;
}